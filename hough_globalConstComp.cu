#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : make
 ============================================================================
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include "common/pgm.h"
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>

const int degreeInc = 2;
const int degreeBins = 180 / degreeInc;
const int rBins = 100;
const float radInc = degreeInc * M_PI / 180;
//*****************************************************************
// The CPU function returns a pointer to the accummulator
void CPU_HoughTran (unsigned char *pic, int w, int h, int **acc)
{
  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;  //(w^2 + h^2)/2, radio max equivalente a centro -> esquina
  *acc = new int[rBins * degreeBins];            //el acumulador, conteo depixeles encontrados, 90*180/degInc = 9000
  memset (*acc, 0, sizeof (int) * rBins * degreeBins); //init en ceros
  int xCent = w / 2;
  int yCent = h / 2;
  float rScale = 2 * rMax / rBins;

  for (int i = 0; i < w; i++) //por cada pixel
    for (int j = 0; j < h; j++) //...
      {
        int idx = j * w + i;
        if (pic[idx] > 0) //si pasa thresh, entonces lo marca
          {
            int xCoord = i - xCent;
            int yCoord = yCent - j;  // y-coord has to be reversed
            float theta = 0;         // actual angle
            for (int tIdx = 0; tIdx < degreeBins; tIdx++) //add 1 to all lines in that pixel
              {
                float r = xCoord * cos (theta) + yCoord * sin (theta);
                int rIdx = (r + rMax) / rScale;
                (*acc)[rIdx * degreeBins + tIdx]++; //+1 para este radio r y este theta
                theta += radInc;
              }
          }
      }
}

//*****************************************************************
// TODO usar memoria constante para la tabla de senos y cosenos
// inicializarlo en main y pasarlo al device
__constant__ float d_Cos[degreeBins];
__constant__ float d_Sin[degreeBins];


//*****************************************************************
//TODO Kernel memoria compartida
// __global__ void GPU_HoughTranShared(...)
// {
//   //TODO
// }
//TODO Kernel memoria Constante
// __global__ void GPU_HoughTranConst(...)
// {
//   //TODO
// }

// GPU kernel. One thread per image pixel is spawned.
// The accummulator memory needs to be allocated by the host in global memory
__global__ void GPU_HoughTran(unsigned char *pic, int w, int h, int *acc, float rMax, float rScale) {
  int locID = threadIdx.x; // ID del hilo local dentro del bloque
  int gloID = blockIdx.x * blockDim.x + locID;

  // Memoria compartida para el acumulador local
  __shared__ int localAcc[degreeBins * rBins];

  // Inicialización del acumulador local
  for (int i = locID; i < degreeBins * rBins; i += blockDim.x) {
    localAcc[i] = 0;
  }

  // Sincronización de los hilos para asegurar que todos hayan completado la inicialización
  __syncthreads();

  if (gloID < w * h) {
    int xCent = w / 2;
    int yCent = h / 2;

    // Calcula las coordenadas del píxel con respecto al centro de la imagen
    int xCoord = (gloID % w) - xCent;
    int yCoord = yCent - (gloID / w); // Invierte la coordenada y debido a que el origen de la imagen está en la parte superior izquierda

    if (pic[gloID] > 0) { // Si el pixel no es negro (más que el umbral)
      for (int tIdx = 0; tIdx < degreeBins; tIdx++) {
        float theta = tIdx * radInc; // Calcula el ángulo actual
        float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
        int rIdx = (int)((r + rMax) / rScale);
        if (rIdx >= 0 && rIdx < rBins) { // Asegúrate de que rIdx esté dentro del rango de rBins
          atomicAdd(&localAcc[rIdx * degreeBins + tIdx], 1);
        }
      }
    }
  }

  // Sincronización de los hilos para asegurar que todos hayan completado la actualización del acumulador local
  __syncthreads();

  // Suma los valores del acumulador local al acumulador global usando un loop
  for (int i = locID; i < degreeBins * rBins; i += blockDim.x) {
    atomicAdd(&acc[i], localAcc[i]);
  }
}

  //TODO eventualmente cuando se tenga memoria compartida, copiar del local al global
  //utilizar operaciones atomicas para seguridad
  //faltara sincronizar los hilos del bloque en algunos lados



// Función para dibujar las líneas detectadas en la imagen original y guardarla
void drawAndSaveLines(const char *outputFileName, unsigned char *originalImage, int w, int h, int *h_hough, float rScale, float rMax, int maxLinesToDraw) {
  cv::Mat img(h, w, CV_8UC1, originalImage);
  cv::Mat imgColor;
  cvtColor(img, imgColor, cv::COLOR_GRAY2BGR);
  int xCent = w / 2;
  int yCent = h / 2;

  // Vector para almacenar las líneas junto con su peso
  std::vector<std::pair<cv::Vec2f, int>> linesWithWeights;

  // Llenar el vector con las líneas y sus pesos
  for (int rIdx = 0; rIdx < rBins; rIdx++) {
    for (int tIdx = 0; tIdx < degreeBins; tIdx++) {
      int weight = h_hough[rIdx * degreeBins + tIdx];
      if (weight > 0) {
        float r = (rIdx * rScale) - (rBins * rScale) / 2;
        float theta = tIdx * radInc;
        linesWithWeights.push_back(std::make_pair(cv::Vec2f(theta, r), weight));
      }
    }
  }

  // Ordenar las líneas por peso en orden descendente
  std::sort(linesWithWeights.begin(), linesWithWeights.end(),
            [](const std::pair<cv::Vec2f, int> &a, const std::pair<cv::Vec2f, int> &b) {
              return a.second > b.second;
            });

  // Dibujar las primeras N líneas (las más fuertes)
  for (int i = 0; i < std::min(maxLinesToDraw, static_cast<int>(linesWithWeights.size())); ++i) {
    cv::Vec2f lineParams = linesWithWeights[i].first;
    float theta = lineParams[0];
    float r = lineParams[1];

    double cosTheta = cos(theta);
    double sinTheta = sin(theta);

    double x0 = xCent - (r * cosTheta);
    double y0 = yCent + (r * sinTheta);  // Note el cambio de signo aquí
    double alpha = sqrt(w * w + h * h);  // Asegura que alpha sea suficientemente grande

    // Puntos de inicio y final para la línea extendida
    cv::Point pt1, pt2;
    pt1.x = cvRound(x0 + alpha * (-sinTheta));
    pt1.y = cvRound(y0 + alpha * cosTheta);
    pt2.x = cvRound(x0 - alpha * (-sinTheta));
    pt2.y = cvRound(y0 - alpha * cosTheta);

    // Dibuja la línea en la imagen
    cv::line(imgColor, pt1, pt2, cv::Scalar(0, 0, 255), 2, cv::LINE_AA);
  }

  // Guardar la imagen con líneas detectadas
  cv::imwrite(outputFileName, imgColor);
}


//*****************************************************************
int main (int argc, char **argv)
{
  int i;

  PGMImage inImg (argv[1]);

  // Declaración de eventos CUDA
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int *cpuht;
  int w = inImg.x_dim;
  int h = inImg.y_dim;

  float* d_Cos;
  float* d_Sin;

  hipMalloc ((void **) &d_Cos, sizeof (float) * degreeBins);
  hipMalloc ((void **) &d_Sin, sizeof (float) * degreeBins);

  // CPU calculation
  CPU_HoughTran(inImg.pixels, w, h, &cpuht);

  // pre-compute values to be stored
  float *pcCos = (float *) malloc (sizeof (float) * degreeBins);
  float *pcSin = (float *) malloc (sizeof (float) * degreeBins);
  float rad = 0;
  for (i = 0; i < degreeBins; i++)
  {
    pcCos[i] = cos (rad);
    pcSin[i] = sin (rad);
    rad += radInc;
  }

  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;
  float rScale = 2 * rMax / rBins;

  // TODO eventualmente volver memoria global
  hipMemcpyToSymbol(HIP_SYMBOL(d_Cos), pcCos, sizeof (float) * degreeBins);
  hipMemcpyToSymbol(HIP_SYMBOL(d_Sin), pcSin, sizeof (float) * degreeBins);

  // setup and copy data from host to device
  unsigned char *d_in, *h_in;
  int *d_hough, *h_hough;

  h_in = inImg.pixels; // h_in contiene los pixeles de la imagen

  h_hough = (int *) malloc (degreeBins * rBins * sizeof (int));

  hipMalloc ((void **) &d_in, sizeof (unsigned char) * w * h);
  hipMalloc ((void **) &d_hough, sizeof (int) * degreeBins * rBins);
  hipMemcpy (d_in, h_in, sizeof (unsigned char) * w * h, hipMemcpyHostToDevice);
  hipMemset (d_hough, 0, sizeof (int) * degreeBins * rBins);

  // Registra el evento de inicio
  hipEventRecord(start);

  // execution configuration uses a 1-D grid of 1-D blocks, each made of 256 threads
  //1 thread por pixel
  int blockNum = ceil (w * h / 256);
  GPU_HoughTran <<< blockNum, 256 >>> (d_in, w, h, d_hough, rMax, rScale);

  // Registra el evento de finalización
  hipEventRecord(stop);

  // Sincroniza el dispositivo para asegurar que el kernel ha terminado
  hipDeviceSynchronize();

  // Calcula el tiempo transcurrido
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  // get results from device
  hipMemcpy (h_hough, d_hough, sizeof (int) * degreeBins * rBins, hipMemcpyDeviceToHost);

  // compare CPU and GPU results
  // for (i = 0; i < degreeBins * rBins; i++)
  // {
  //   if (cpuht[i] != h_hough[i])
  //     printf ("Calculation mismatch at : %i %i %i\n", i, cpuht[i], h_hough[i]);
  // }
  printf("Done!\n");

  // Imprime el tiempo transcurrido
  printf("Tiempo transcurrido: %f seg\n", milliseconds / 1000);

  // Draw and save lines on the original image
  drawAndSaveLines("output_image_globalConstCompu.jpg", inImg.pixels, w, h, h_hough, rScale, rMax, 40);

  // TODO clean-up
  hipFree(d_in);
  hipFree(d_hough);
  hipFree(d_Cos);
  hipFree(d_Sin);
  free(pcCos);
  free(pcSin);
  free(h_hough);

  // Destruye los eventos CUDA
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
